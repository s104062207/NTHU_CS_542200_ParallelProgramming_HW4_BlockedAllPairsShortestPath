#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <omp.h>
struct timespec s, e, all_start, all_end;
long cpu_time = 0, io_time = 0, communication_time = 0;
void calc_time(long* target, struct timespec a, struct timespec b)
{
    int sec = a.tv_sec - b.tv_sec;
    int nsec = a.tv_nsec - b.tv_nsec;
    *target += ((long) sec) * 1000000000 + nsec;
}
__global__ 
void calPhase1(int B, int Round, int* Dist, int node, int pitch)
{
	extern __shared__ int sdata[];
	int x = threadIdx.x;
    int y = threadIdx.y;
	int sx = Round*B+x;
	int sy = Round*B+y;
	
	sdata[x*B+y]=Dist[sx*pitch+sy];
	__syncthreads();
	int tem;
	#pragma unroll
	for (int k = 0; k < B ; ++k) 
	{		
		tem=sdata[x*B+k] + sdata[k*B+y];
		if (tem < sdata[x*B+y])
			sdata[x*B+y] = tem;	
		__syncthreads();
	}
	Dist[sx*pitch+sy]=sdata[x*B+y];
	__syncthreads();
}
__global__ 
void calPhase2(int B, int Round, int* Dist, int node, int pitch)
{
	if(blockIdx.x==Round)
		return;
	extern __shared__ int sm[];
	int* p = &sm[B*B];
	
	int x = threadIdx.x;
    int y = threadIdx.y;
	
	unsigned int sx = Round*B+x;
	unsigned int sy = Round*B+y;	
	sm[x*B+y]=Dist[sx*pitch+sy];
	
	unsigned int rx = blockIdx.x*B+x;
	unsigned int cy = blockIdx.x*B+y;
	unsigned int idx= (blockIdx.y == 1)?rx*pitch+sy:sx*pitch+cy;
	p[x*B+y]=Dist[idx];
	__syncthreads();
	
	int* a =(blockIdx.y == 0)?&sm[0]:p;
	int* b =(blockIdx.y == 1)?&sm[0]:p;
	int tem;
	#pragma unroll
	for (int k = 0; k < B ; ++k) 
	{
		tem=a[x*B+k] + b[k*B+y];
		if ( tem < p[x*B+y])
			p[x*B+y] = tem;
	}
	Dist[idx]=p[x*B+y];
}
__global__ 
void calPhase3(int B, int Round, int* Dist, int node, int pitch,int threadId,int halfRound)
{
	int blockIdxx=blockIdx.x;
	if(threadId==1)
		blockIdxx=blockIdxx+halfRound;
	if (blockIdxx == Round || blockIdx.y == Round) 
		return;
	extern __shared__ int sm[];
	int* pr = &sm[0];
	int* pc = &sm[B*B];
	
	int x = threadIdx.x;
    int y = threadIdx.y;
	
	int rx = blockIdxx*blockDim.x+x;
	int ry = Round*B+y;
	
	int cx = Round*B+x;
	int cy = blockIdx.y*blockDim.y+y;
	
	pr[x*B+y]=Dist[rx*pitch+ry];
	pc[x*B+y]=Dist[cx*pitch+cy];
	__syncthreads();
	
	if (rx >= node || cy >= node) 
		return;
	
	int tem;
	int ans=Dist[rx*pitch+cy] ;
	#pragma omp parallel for 
	for (int k = 0; k < B ; ++k) {		
		tem=pr[x*B+k] + pc[k*B+y];
		if ( tem<ans){
			ans=tem;
		}
	}
	Dist[rx*pitch+cy] = ans;
}
int main(int argc, char* argv[])
{
	
	
	//input(argv[1]);
	FILE *fp = fopen(argv[1], "rb");
	int n, edge;
	clock_gettime(CLOCK_REALTIME, &s);
	fread(&n   , sizeof(int), 1, fp); 
	fread(&edge, sizeof(int), 1, fp);
	clock_gettime(CLOCK_REALTIME, &e);
    calc_time(&io_time, e, s);
	
	int B = (n>32)?32:16;//atoi(argv[3]);
	int round = (n + B -1)/B;
	int pitch_n = round*B;//(n%B==0)?n:n-n%B+B;
	int* Dist;//=(int*) malloc(pitch_n * pitch_n * sizeof(int));
    hipHostMalloc(&Dist, sizeof(int)*pitch_n*pitch_n);
	clock_gettime(CLOCK_REALTIME, &s);
	#pragma omp parallel for collapse(2)
	for (int i = 0; i < pitch_n; ++i) {
		for (int j = 0; j < pitch_n; ++j) {
			if (i == j)	
				Dist[i*pitch_n+j] = 0;
			else		
				Dist[i*pitch_n+j] = 1000000000;
		}
	}
	    clock_gettime(CLOCK_REALTIME, &e);
    calc_time(&cpu_time, e, s);
	int* temp =(int*) malloc(edge * 3 * sizeof(int));
	clock_gettime(CLOCK_REALTIME, &s);
    fread(temp, sizeof(int), edge * 3, fp);
	    clock_gettime(CLOCK_REALTIME, &e);
    calc_time(&io_time, e, s);
	
	
	clock_gettime(CLOCK_REALTIME, &s);
    #pragma omp parallel for 
    for (int i = 0; i < edge*3; i=i+3) 
        Dist[temp[i]*pitch_n+temp[i+1]] = temp[i+2];
	    clock_gettime(CLOCK_REALTIME, &e);
    calc_time(&cpu_time, e, s);
	
	//block_FW(B);
	float time;
    float GPU_time = 0;
    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
	int* devDist[2];
	#pragma omp parallel num_threads(2)
    {
		int threadId = omp_get_thread_num();
		//hipSetDevice(threadId);
		hipMalloc(&devDist[threadId], sizeof(int) * pitch_n * pitch_n);
		hipMemcpy(devDist[threadId], Dist, sizeof(int) * pitch_n * pitch_n, hipMemcpyHostToDevice);
	
		dim3 grid1(1, 1);
		dim3 grid2(round, 2);
		dim3 grid3(round, round);
		
		if(threadId == 0) grid3.x = round/2;
		else grid3.x = round-(round/2);
		
		dim3 block(B, B);
		int sSize = B * B * sizeof(int);
		hipEventRecord (start, 0);
		for (int r = 0; r < round; ++r) {
			calPhase1<<<grid1, block, sSize  >>>(B, r, devDist[threadId], n, pitch_n);
			calPhase2<<<grid2, block, sSize*2>>>(B, r, devDist[threadId], n, pitch_n);
			calPhase3<<<grid3, block, sSize*2>>>(B, r, devDist[threadId], n, pitch_n,threadId,round/2);
			hipDeviceSynchronize();
			clock_gettime(CLOCK_REALTIME, &s);
			#pragma omp barrier
			if(threadId==0)
				hipMemcpyPeer(devDist[1], 1, devDist[0], 0, round/2*B*sizeof(int) * pitch_n);
			else
				hipMemcpyPeer(&devDist[0][round/2 *pitch_n*B], 0, &devDist[1][round/2 *pitch_n*B], 1, (round-round/2)*B*sizeof(int) * pitch_n);
			#pragma omp barrier
			clock_gettime(CLOCK_REALTIME, &e);
            calc_time(&communication_time, e, s);
		}
		
	}
	hipEventRecord (stop, 0);
	hipEventElapsedTime (&time, start, stop);
	GPU_time = time/1000 - (communication_time/1000000000.0);
	hipDeviceSynchronize();
	hipMemcpy2D(Dist, sizeof(int) *n, devDist[0], sizeof(int) * pitch_n, sizeof(int) *n, n, hipMemcpyDeviceToHost);
	
	//output(argv[2]);
	fp = fopen(argv[2], "wb+");
	clock_gettime(CLOCK_REALTIME, &s);
	fwrite(Dist, sizeof(int), n*n, fp);
	clock_gettime(CLOCK_REALTIME, &e);
	calc_time(&io_time, e, s);
	
	printf("io_time:%lf\n",(io_time/1000000000.0));
	printf ("GPU time = %lf\n", GPU_time);
	printf("communication time = %lf\n",(communication_time/1000000000.0));
	printf("cpu_time:%lf\n",(cpu_time/1000000000.0));
	fclose(fp);

	return 0;
}
